#include "hip/hip_runtime.h"
#define __USE_CUDA_HEAD__
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

typedef unsigned long DWORD;
typedef struct GraphicBasicPoint_
{
	//store value for one frame
public:
	float x;
	float y;
	float width;
	float glowWidth;
	float shadowWidth;
	DWORD color;
	DWORD glowColor;
	DWORD shadowColor;
} GraphicBasicPoint;

typedef struct CUDARenderSetting_
{
	//store value for one frame
public:
	float camX;
	float camY;
	float scaleX;
	float scaleY;
	int sizeX;
	int sizeY;
	int graphicCount;
} CUDARenderSetting;

#include "head.cu"

__global__ void transformKernel(
	CUDARenderSetting * setting_dev,
	GraphicBasicPoint * points_dev
)
{
	float x = setting_dev->camX;
	float y = setting_dev->camY;
	float scaleX = setting_dev->scaleX;
	float scaleY = setting_dev->scaleY;

	points_dev[blockIdx.x].x -= x;
	points_dev[blockIdx.x].y -= y;
	points_dev[blockIdx.x].x *+ scaleX;
	points_dev[blockIdx.x].y *= scaleY;
}

__global__ void renderKernel(
	CUDARenderSetting * setting_dev,
	GraphicBasicPoint * points_dev,
	DWORD * fillColor_dev,
	int * startPos_dev,
	DWORD * res_dev
)
{

}


bool CUDARenderCore::init()
{
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return false;
	}

	return true;
}



bool CUDARenderCore::render(DWORD * output,
	int height, int width,
	float camX, float camY,
	float scaleX, float scaleY,
	int graphicCount,
	int pointCount,
	GraphicBasicPoint * points,
	DWORD * fillColor,
	int * startPos
)
{
	hipError_t cudaStatus;

	CUDARenderSetting setting;
	setting.camX = camX;
	setting.camY = camY;
	setting.graphicCount = graphicCount;
	setting.scaleX = scaleX;
	setting.scaleY = scaleY;
	setting.sizeX = width;
	setting.sizeY = height;

	CUDARenderSetting * setting_dev;
	GraphicBasicPoint * points_dev;
	DWORD * fillColor_dev;
	int * startPos_dev;
	DWORD * res_dev;

	cudaStatus = hipMalloc((void**)&setting_dev, sizeof(CUDARenderSetting));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return false;
	}

	cudaStatus = hipMalloc((void**)&points_dev, (pointCount + 16) * sizeof(GraphicBasicPoint));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return false;
	}

	cudaStatus = hipMalloc((void**)&fillColor_dev, (graphicCount + 16) * sizeof(DWORD));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return false;
	}

	cudaStatus = hipMalloc((void**)&startPos_dev, (graphicCount + 16) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return false;
	}

	cudaStatus = hipMalloc((void**)&res_dev, height * width * sizeof(DWORD));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return false;
	}

	cudaStatus = hipMemcpy(setting_dev, &setting, sizeof(CUDARenderSetting), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}

	cudaStatus = hipMemcpy(points_dev, points, pointCount * sizeof(GraphicBasicPoint), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}

	cudaStatus = hipMemcpy(fillColor_dev, fillColor, graphicCount * sizeof(DWORD), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}


	cudaStatus = hipMemcpy(startPos_dev, startPos, graphicCount * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}

	transformKernel <<<graphicCount, 1 >>> (setting_dev, points_dev);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "transform Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	dim3 blocks(width, height);
	renderKernel <<<blocks, 1 >>> (setting_dev, points_dev, fillColor_dev, startPos_dev, res_dev);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "render Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, res_dev, height * width * sizeof(DWORD), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


Error:
	hipFree(setting_dev);
	hipFree(points_dev);
	hipFree(fillColor_dev);
	hipFree(startPos_dev);
	hipFree(res_dev);

	return cudaStatus;

}