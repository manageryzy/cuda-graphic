#include "hip/hip_runtime.h"
#define __USE_CUDA_HEAD__
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

typedef unsigned long DWORD;
typedef struct GraphicBasicPoint_
{
	//store value for one frame
public:
	float x;
	float y;
	float width;
	float glowWidth;
	float shadowWidth;
	DWORD color;
	DWORD glowColor;
	DWORD shadowColor;
} GraphicBasicPoint;

typedef struct CUDARenderSetting_
{
	//store value for one frame
public:
	float camX;
	float camY;
	float scaleX;
	float scaleY;
	int sizeX;
	int sizeY;
	int graphicCount;
} CUDARenderSetting;

#include "head.cu"

__global__ void transformKernel(
	CUDARenderSetting * setting_dev,
	GraphicBasicPoint * points_dev
);

__global__ void renderKernel(
	CUDARenderSetting * setting_dev,
	GraphicBasicPoint * points_dev,
	DWORD * fillColor_dev,
	int * startPos_dev,
	DWORD * res_dev
);

DWORD inline __device__  blendColor(DWORD color1, DWORD color2);
DWORD inline __device__  gradientColor(DWORD color1, DWORD color2, float s);
float inline __device__  gradientFloat(float x, float y, float s);
float inline __device__  pointToSegDist(float x, float y, float x1, float y1, float x2, float y2);



__global__ void transformKernel(
	CUDARenderSetting * setting_dev,
	GraphicBasicPoint * points_dev
)
{
	float x = setting_dev->camX;
	float y = setting_dev->camY;
	float scaleX = setting_dev->scaleX;
	float scaleY = setting_dev->scaleY;

	points_dev[blockIdx.x].x -= x;
	points_dev[blockIdx.x].y -= y;
	points_dev[blockIdx.x].x *= scaleX;
	points_dev[blockIdx.x].y *= scaleY;
}

__global__ void renderKernel(
	CUDARenderSetting * setting_dev,
	GraphicBasicPoint * points_dev,
	DWORD * fillColor_dev,
	int * startPos_dev,
	DWORD * res_dev
)
{
	DWORD color = 0;

	float x = blockIdx.x;
	float y = blockIdx.y;

	for (int i = 0; i < setting_dev->graphicCount; i++)
	{
		for (int j = startPos_dev[i]; j < startPos_dev[i + 1] - 1; j++)
		{
			float x1 = points_dev[j].x;
			float y1 = points_dev[j].y;
			float x2 = points_dev[j + 1].x;
			float y2 = points_dev[j + 1].y;

			float distToSeg;
			float r;
			{
				float cross = (x2 - x1) * (x - x1) + (y2 - y1) * (y - y1);
				float d2 = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1);
				r = cross / d2;
				
				if (cross <= 0)
				{
					distToSeg = sqrtf((x - x1) * (x - x1) + (y - y1) * (y - y1));
				}
				else if (cross >= d2)
				{
					distToSeg = sqrtf((x - x2) * (x - x2) + (y - y2) * (y - y2));

				}
				else
				{
					float px = x1 + (x2 - x1) * r;
					float py = y1 + (y2 - y1) * r;
					distToSeg = sqrtf((x - px) * (x - px) + (y - py) * (y - py));
				}
			}

			float point_width = gradientFloat(points_dev[j].width, points_dev[j + 1].width,r)*setting_dev->scaleX/2;

			if (distToSeg <= point_width  )
			{
				DWORD point_color = gradientColor(points_dev[j].color, points_dev[j + 1].color, r);
				color = blendColor(point_color, color);
			}
			
		}
	}
	
	res_dev[blockIdx.y * setting_dev->sizeX + blockIdx.x] = color;
	
	//res_dev[blockIdx.y * setting_dev->sizeX + blockIdx.x] = (a << 24)| (b << 16) |(g << 8) |r;
}

DWORD inline __device__  blendColor(DWORD color1,DWORD color2)
{
	unsigned char r1 = color1 & 0xFF;
	unsigned char g1 = (color1 >> 8) & 0xFF;
	unsigned char b1 = (color1 >> 16) & 0xFF;
	unsigned char a1 = (color1 >> 24) & 0xFF;
	unsigned char r2 = color2 & 0xFF;
	unsigned char g2 = (color2 >> 8) & 0xFF;
	unsigned char b2 = (color2 >> 16) & 0xFF;
	unsigned char a2 = (color2 >> 24) & 0xFF;

	unsigned char r = (r1*a1 + r2*a2) / (a1 + a2);
	unsigned char g = (g1*a1 + g2*a2) / (a1 + a2);
	unsigned char b = (b1*a1 + b2*a2) / (a1 + a2);

	return (0xFF << 24) | (b << 16) | (g << 8) | r;
}

DWORD inline __device__  gradientColor(DWORD color1, DWORD color2, float s)
{
	float s_ = 1 - s;
	int r1 = color1 & 0xFF;
	int g1 = (color1 >> 8) & 0xFF;
	int b1 = (color1 >> 16) & 0xFF;
	int a1 = (color1 >> 24) & 0xFF;
	int r2 = color2 & 0xFF;
	int g2 = (color2 >> 8) & 0xFF;
	int b2 = (color2 >> 16) & 0xFF;
	int a2 = (color2 >> 24) & 0xFF;

	unsigned char r = (r1*s + r2*s_);
	unsigned char g = (g1*s + g2*s_);
	unsigned char b = (b1*s + b2*s_);
	unsigned char a = (a1*s + a2*s_);

	return (a << 24) | (b << 16) | (g << 8) | r;
}

float inline __device__  pointToSegDist(float x, float y, float x1, float y1, float x2, float y2)
{
	float cross = (x2 - x1) * (x - x1) + (y2 - y1) * (y - y1);
	if (cross <= 0) return sqrtf((x - x1) * (x - x1) + (y - y1) * (y - y1));

	float d2 = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1);
	if (cross >= d2) return sqrtf((x - x2) * (x - x2) + (y - y2) * (y - y2));

	float r = cross / d2;
	float px = x1 + (x2 - x1) * r;
	float py = y1 + (y2 - y1) * r;

	return sqrtf((x - px) * (x - px) + (y - py) * (y - py));
}

float inline __device__  gradientFloat(float x, float y, float s)
{
	return x*s + y*(1 - s);
}


bool CUDARenderCore::init()
{
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return false;
	}

	return true;
}



bool CUDARenderCore::render(DWORD * output,
	int height, int width,
	float camX, float camY,
	float scaleX, float scaleY,
	int graphicCount,
	int pointCount,
	GraphicBasicPoint * points,
	DWORD * fillColor,
	int * startPos
)
{
	hipError_t cudaStatus;

	CUDARenderSetting setting;
	setting.camX = camX;
	setting.camY = camY;
	setting.graphicCount = graphicCount;
	setting.scaleX = scaleX;
	setting.scaleY = scaleY;
	setting.sizeX = width;
	setting.sizeY = height;

	CUDARenderSetting * setting_dev;
	GraphicBasicPoint * points_dev;
	DWORD * fillColor_dev;
	int * startPos_dev;
	DWORD * res_dev;

	cudaStatus = hipMalloc((void**)&setting_dev, sizeof(CUDARenderSetting));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return false;
	}

	cudaStatus = hipMalloc((void**)&points_dev, (pointCount + 16) * sizeof(GraphicBasicPoint));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return false;
	}

	cudaStatus = hipMalloc((void**)&fillColor_dev, (graphicCount + 16) * sizeof(DWORD));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return false;
	}

	cudaStatus = hipMalloc((void**)&startPos_dev, (graphicCount + 16) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return false;
	}

	cudaStatus = hipMalloc((void**)&res_dev, height * width * sizeof(DWORD));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return false;
	}

	cudaStatus = hipMemcpy(setting_dev, &setting, sizeof(CUDARenderSetting), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}

	cudaStatus = hipMemcpy(points_dev, points, (pointCount+16) * sizeof(GraphicBasicPoint), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}

	cudaStatus = hipMemcpy(fillColor_dev, fillColor, (graphicCount+16) * sizeof(DWORD), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}


	cudaStatus = hipMemcpy(startPos_dev, startPos, (graphicCount+16) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return false;
	}

	if (graphicCount != 0)
	{
		transformKernel <<<pointCount, 1 >>> (setting_dev, points_dev);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "transform Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
	}

	dim3 blocks(width, height);
	renderKernel <<<blocks, 1 >>> (setting_dev, points_dev, fillColor_dev, startPos_dev, res_dev);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "render Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, res_dev, height * width * sizeof(DWORD), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


Error:
	hipFree(setting_dev);
	hipFree(points_dev);
	hipFree(fillColor_dev);
	hipFree(startPos_dev);
	hipFree(res_dev);

	return cudaStatus;

}