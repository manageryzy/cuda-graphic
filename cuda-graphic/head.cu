#pragma once
#include <hip/hip_runtime.h>

#ifdef __USE_CUDA_HEAD__
namespace CUDARenderCore
{
	bool init();
	cudaError_t render(DWORD * output,
		int height, int width, 
		float camX, float camY, 
		float scaleX, float scaleY,
		int graphicCount,
		int pointCount,
		GraphicBasicPoint * points,
		DWORD * fillColor,
		int * startPos
	);
}
#endif